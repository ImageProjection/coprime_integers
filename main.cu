#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <climits>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

#define N_batches 100 //Matrix size = N_batches*batch_size
#define batch_size 1024

__global__ fill_matr(int* d_matr, int matrix_size)
{
	int idx=blockIdx.x*blockDim.x + threadIdx.x;
	int idy=blockIdx.y*blockDim.y + threadIdx.y;
	for(int i=0;i<N_batches;i++)//find gcd of elements in the batch, then move on to next
	{
		//d_matr[idx][idy]=gcd(idx,idy)
		//idx+=batch_size
		//idy+=batch_size
		
	}
}

int main()
{
    clock_t start,end;
	start=clock();

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("kernel timeout enabled: %d\n",prop.kernelExecTimeoutEnabled);

	const int matrix_size=N_batches*batch_size;
	//files
	FILE *out_matr;
	out_matr=fopen("out_matr.txt","w");
	//matr 1d array
	int* d_matr;
	hipMalloc((void**)&d_matr, matrix_size*sizeof(int));
	int* h_matr;
	h_matr=(int*)malloc(matrix_size*sizeof(int));
	
	//kernel launch config
	dim3 grid_conf(matrix_size,1,1);
	dim3 block_conf(1,batch_size,1);

	fill_matr<<<grid_conf,block_conf>>>(d_matr, matrix_size);

	hipMemcpy(h_matr, d_matr, matrix_size*sizeof(int), hipMemcpyDeviceToHost);

	print_matr(h_matr,matrix_size);

	hipFree(d_matr);
	free(h_matr);
	fclose(out_matr);

	printf("===launch status report===\n");
	//check for errors
	hipError_t err=hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA ERROR!!!\n");
		printf("err code: %d\n",err);
		if (err == 702)
		{
			printf("702 is similar to WDDM TDR false trigger; suggest running from tty3\n");
		}
		if (err == 700)
		{
			printf("700 is out of range call\n");
		}
	}
	else
	{
		printf("No CUDA errors!!!\n");
	}

	end=clock();
	double total_time=(double)(end-start)/CLOCKS_PER_SEC;//in seconds
	printf("TOTAL TIME: %.1lf seconds (%.1lf minutes)\n",total_time,total_time/60);
}
